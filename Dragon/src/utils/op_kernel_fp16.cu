
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include <cmath>

#include "core/context_cuda.h"
#include "core/tensor.h"
#include "utils/cuda_device.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"
#include "utils/cast.h"

namespace dragon {

namespace kernel {

template <typename T>
__global__ void _EmptyHalf() {}

template<> void Empty<float16, CUDAContext>() {
    _EmptyHalf<float16> << <1, 1 >> >();
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.relu ********************/

#ifdef WITH_CUDA_FP16
template <typename T>
__global__ void _ReluHalf(const int count, const half* x, const half slope, half* y) {
    const half kZero = __float2half(0.f);
    CUDA_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        y[idx] = __hgt(x[idx], kZero) ? x[idx] : __hmul(x[idx], slope);
#endif
    }
}

template <typename T>
__global__ void _ReluHalf2(const int count, const half2* x, const half2 slope, half2* y) {
    const half2 kZero = __float2half2_rn(0.f);
    CUDA_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        y[idx] = __hbgt2(x[idx], kZero) ? x[idx] : __hmul2(x[idx], slope);
#endif
    }
}
#endif

template<> void Relu<float16, CUDAContext>(const int count,
                                           const float16* x,
                                           const float slope,
                                           float16* y) {
#ifdef WITH_CUDA_FP16
    if (count % 2 == 0) 
        _ReluHalf2<half2> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> > (count / 2,
                                                 reinterpret_cast<const half2*>(x),
                                                  dragon_cast<half2, float>(slope),
                                                      reinterpret_cast<half2*>(y));
    else
        _ReluHalf<half> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                           reinterpret_cast<const half*>(x),
                                            dragon_cast<half, float>(slope),
                                                reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

/******************** arithmetic.affine ********************/

#ifdef WITH_CUDA_FP16
template <typename T>
__global__ void _AffineWithOBiasHalf(const int count,
                                     const int scale_dim,
                                     const int inner_dim,
                                     const half* x,
                                     const half* alpha,
                                     half* y) {
    CUDA_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hmul(alpha[scale_idx], x[idx]);
#endif
    }
}

template <typename T>
__global__ void _AffineWithBiasHalf(const int count,
                                    const int scale_dim,
                                    const int inner_dim,
                                    const half* x,
                                    const half* alpha,
                                    const half* beta,
                                    half* y) {
    CUDA_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hadd(__hmul(alpha[scale_idx], x[idx]), beta[scale_idx]);
#endif
    }
}
#endif

template<> void Affine<float16, CUDAContext>(const int count,
                                             const int outer_dim,
                                             const int scale_dim,
                                             const int inner_dim,
                                             const float16* x,
                                             const float16* alpha,
                                             const float16* beta,
                                             const float16* beta_multiplier,
                                             float16* y) {
#ifdef WITH_CUDA_FP16
    if (beta != nullptr) {
        _AffineWithBiasHalf<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                 scale_dim, inner_dim,
                                                     reinterpret_cast<const half*>(x),
                                                 reinterpret_cast<const half*>(alpha),
                                                  reinterpret_cast<const half*>(beta),
                                                          reinterpret_cast<half*>(y));
    } else {
        _AffineWithOBiasHalf<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                  scale_dim, inner_dim,
                                                      reinterpret_cast<const half*>(x),
                                                  reinterpret_cast<const half*>(alpha),
                                                           reinterpret_cast<half*>(y));
    }
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

/******************** misc.astype ********************/

#ifdef WITH_CUDA_FP16
__global__ void _TypeHalf2Float(const int count, const half* a, float* b) {
    CUDA_KERNEL_LOOP(idx, count) {
        b[idx] = __half2float(a[idx]);
    }
}
__global__ void _TypeFloat2Half(const int count, const float* a, half* b) {
    CUDA_KERNEL_LOOP(idx, count) {
        b[idx] = __float2half(a[idx]);
    }
}

__global__ void _TypeHalf2Half(const int count, const half* a, half* b) {
    CUDA_KERNEL_LOOP(idx, count) {
        b[idx] = a[idx];
    }
}
#endif

#define DEFINE_TYPE_DISABLE_FP16(type) \
    template <> void TypeA2B<float16, type, CUDAContext>(const int count, \
                                                         const float16* a, \
                                                         type* b) { \
        LOG(FATAL) << "CUDAContext has not implemented: float16 -> " \
                   << TypeMetaToString(TypeMeta::Make<type>()); \
    } \
    template <> void TypeA2B<type, float16, CUDAContext>(const int count, \
                                                         const type* a, \
                                                         float16* b) { \
        LOG(FATAL) << "CUDAContext has not implemented: " \
                   << TypeMetaToString(TypeMeta::Make<type>()) << " -> float16"; \
    }

#define DEFINE_TYPE_ENABLE_FP16_FP32 \
    template <> void TypeA2B<float16, float, CUDAContext>(const int count, \
                                                          const float16* a, \
                                                          float* b) { \
        _TypeHalf2Float << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, \
                                      reinterpret_cast<const half*>(a), b); \
        CUDA_POST_KERNEL_CHECK; \
    } \
    template <> void TypeA2B<float, float16, CUDAContext>(const int count, \
                                                          const float* a, \
                                                          float16* b) { \
        _TypeFloat2Half << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, \
                                            a, reinterpret_cast<half*>(b)); \
        CUDA_POST_KERNEL_CHECK; \
    }

#ifdef WITH_CUDA_FP16
template <> void TypeA2B<float16, float16, CUDAContext>(const int count,
                                                        const float16* a,
                                                        float16* b) {
    _TypeHalf2Half << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                     reinterpret_cast<const half*>(a),
                                          reinterpret_cast<half*>(b));
    CUDA_POST_KERNEL_CHECK;
}
DEFINE_TYPE_ENABLE_FP16_FP32;
DEFINE_TYPE_DISABLE_FP16(double);
DEFINE_TYPE_DISABLE_FP16(int);
DEFINE_TYPE_DISABLE_FP16(int64_t);
DEFINE_TYPE_DISABLE_FP16(uint8_t);
#else
template <> void TypeA2B<float16, float16, CUDAContext>(const int count,
                                                        const float16* a,
                                                        float16* b) {
    LOG(FATAL) << "CUDAContext has not implemented: float16 -> float16";
}
DEFINE_TYPE_DISABLE_FP16(float);
DEFINE_TYPE_DISABLE_FP16(double);
DEFINE_TYPE_DISABLE_FP16(int);
DEFINE_TYPE_DISABLE_FP16(int64_t);
DEFINE_TYPE_DISABLE_FP16(uint8_t);
#endif

/******************** misc.image_data ********************/

#ifdef WITH_CUDA_FP16
template <typename Tx, typename Ty>
__global__ void _ImageDataHalf_NCHW(const int count,
                                    const int N, const int C,
                                    const int H, const int W,
                                    const float* mean_values,
                                    const float* std_values,
                                    const Tx* x, 
                                    Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % W;
        const int h = (idx / W) % H;
        const int c = (idx / W / H) % C;
        const int n = idx / W / H / C;
        float raw_value = x[((n * H + h) * W + w) * C + c];
        if (mean_values != nullptr) raw_value -= mean_values[c];
        if (std_values != nullptr) raw_value /= std_values[c];
        y[idx] = __float2half(raw_value);
    }
}

template <typename Tx, typename Ty>
__global__ void _ImageDataHalf_NHWC(const int count,
                                    const int N, const int C,
                                    const int H, const int W,
                                    const float* mean_values,
                                    const float* std_values,
                                    const Tx* x, 
                                    Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int c = idx % C;
        float raw_value = x[idx];
        if (mean_values != nullptr) raw_value -= mean_values[c];
        if (std_values != nullptr) raw_value /= std_values[c];
        y[idx] = __float2half(raw_value);
    }
}
#endif

template <> void ImageData<float, float16, CUDAContext>(const int count,
                                                        const int N, const int C,
                                                        const int H, const int W,
                                                        const float* mean_values,
                                                        const float* std_values,
                                                        const string& data_format,
                                                        const float* x,
                                                        float16* y) {
#ifdef WITH_CUDA_FP16
    if (data_format == "NCHW") {
        _ImageDataHalf_NCHW<float, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                                 N, C, H, W,
                                                                                mean_values,
                                                                                 std_values,
                                                                                          x,
                                                                reinterpret_cast<half*>(y));
    } else if (data_format == "NHWC") {
        _ImageDataHalf_NHWC<float, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                                 N, C, H, W,
                                                                                mean_values,
                                                                                 std_values,
                                                                                          x,
                                                                reinterpret_cast<half*>(y));

    } else LOG(FATAL) << "Unknown data format: " << data_format;
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

template <> void ImageData<uint8_t, float16, CUDAContext>(const int count,
                                                          const int N, const int C,
                                                          const int H, const int W,
                                                          const float* mean_values,
                                                          const float* std_values,
                                                          const string& data_format,
                                                          const uint8_t* x,
                                                          float16* y) {
#ifdef WITH_CUDA_FP16
    if (data_format == "NCHW") {
        _ImageDataHalf_NCHW<uint8_t, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                                   N, C, H, W,
                                                                                  mean_values,
                                                                                   std_values,
                                                                                            x,
                                                                  reinterpret_cast<half*>(y));
    } else if (data_format == "NHWC") {
        _ImageDataHalf_NHWC<uint8_t, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                                   N, C, H, W,
                                                                                  mean_values,
                                                                                   std_values,
                                                                                            x,
                                                                  reinterpret_cast<half*>(y));

    } else LOG(FATAL) << "Unknown data format: " << data_format;
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

/******************** ndarray.concat ********************/

template <typename T>
__global__ void _ConcatHalf(const int count,
                            const int outer_dim,
                            const int inner_dim,
                            const int x_concat_dim,
                            const int y_concat_dim,
                            const int concat_offset,
                            const T* x,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        y[y_idx] = x[idx];
    }
}

template <> void Concat<float16, CUDAContext>(const int count,
                                              const int outer_dim,
                                              const int inner_dim,
                                              const int x_concat_dim,
                                              const int y_concat_dim,
                                              const int concat_offset,
                                              const float16* x,
                                              float16* y,
                                              CUDAContext* context) {
#ifdef WITH_CUDA_FP16
    _ConcatHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                               outer_dim,
                                                               inner_dim,
                                                            x_concat_dim,
                                                            y_concat_dim,
                                                           concat_offset,
                                        reinterpret_cast<const half*>(x),
                                             reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

template <typename T>
__global__ void _ConcatGradHalf(const int count,
                                const int outer_dim,
                                const int inner_dim,
                                const int x_concat_dim,
                                const int y_concat_dim,
                                const int concat_offset,
                                const T* dy,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        dx[idx] = dy[y_idx];
    }
}

template <> void ConcatGrad<float16, CUDAContext>(const int count,
                                                  const int outer_dim,
                                                  const int inner_dim,
                                                  const int x_concat_dim,
                                                  const int y_concat_dim,
                                                  const int concat_offset,
                                                  const float16* dy,
                                                  float16* dx,
                                                  CUDAContext* context) {
#ifdef WITH_CUDA_FP16
    _ConcatGradHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                   outer_dim,
                                                                   inner_dim,
                                                                x_concat_dim,
                                                                y_concat_dim,
                                                               concat_offset,
                                           reinterpret_cast<const half*>(dy),
                                                reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

/******************** ndarray.transpose ********************/

template <typename T>
__global__ void _TransposeHalf(const int count,
                               const int ndim,
                               const int* order,
                               const int* old_steps,
                               const int* new_steps,
                               const T* x,
                               T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
       int x_idx = 0, y_idx = idx;
       for (int j = 0; j < ndim; ++j) {
           int k = order[j];
           x_idx += (y_idx / new_steps[j]) * old_steps[k];
           y_idx %= new_steps[j];
       }
       y[idx] = x[x_idx];
   }
}

template <> void Transpose<float16, CUDAContext>(const int count,
                                                 const int ndim,
                                                 const int* order,
                                                 const int* old_steps,
                                                 const int* new_steps,
                                                 const float16* x,
                                                 float16* y) {
#ifdef WITH_CUDA_FP16
    _TransposeHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                       ndim,
                                                                      order,
                                                                  old_steps,
                                                                  new_steps,
                                           reinterpret_cast<const half*>(x),
                                                reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

template <typename T>
__global__ void _TransposeGradHalf(const int count,
                                   const int ndim,
                                   const int* order,
                                   const int* old_steps,
                                   const int* new_steps,
                                   const T* dy,
                                   T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int x_idx = 0, y_idx = idx;
        for (int j = 0; j < ndim; ++j) {
            int k = order[j];
            x_idx += (y_idx / new_steps[j]) * old_steps[k];
            y_idx %= new_steps[j];
        }
        dx[x_idx] = dy[idx];
    }
}

template <> void TransposeGrad<float16, CUDAContext>(const int count,
                                                     const int ndim,
                                                     const int* order,
                                                     const int* old_steps,
                                                     const int* new_steps,
                                                     const float16* dy,
                                                     float16* dx) {
#ifdef WITH_CUDA_FP16
    _TransposeGradHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                           ndim,
                                                                          order,
                                                                      old_steps,
                                                                      new_steps,
                                              reinterpret_cast<const half*>(dy),
                                                   reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
#else
    CUDA_FP16_NOT_COMPILED;
#endif
}

}    // namespace kernel

}    // namespace dragon

#endif // WITH_CUDA